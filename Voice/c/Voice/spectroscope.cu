#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil.h>

const int THREAD_N = 128;
const int BLOCK_N = 4;

int bufFrameLength;
int freqCount;

__device__ float *d_data;
__device__ float *d_freq;
__device__ int *d_ifreq;

__device__ float *d_xs;
__device__ float *d_ys;
__device__ float *d_as;
__device__ float *d_maxes;

__device__ float *d_periods;
__device__ float *d_rces;
__device__ float *d_rses;
__device__ float *d_dampes;

#define PI 3.14159265358979323846264338327950288f

__global__ void dev_init(int freqCount, 
						 float *d_xs,
						 float *d_ys,
						 float *d_as,
						 float *d_maxes,
						 
						 float *d_periods, 
						 float *d_rces,
						 float *d_rses,
						 float *d_dampes
){
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;
	const int threadN = blockDim.x * gridDim.x;

	for (int f = tid; f<freqCount; f+=threadN){
		d_xs[f] = 0;
		d_ys[f] = 0;
		d_as[f] = 0;
		d_maxes[f] = 0;

		const float period = d_periods[f];
		d_rces[f] = cosf(2*PI/period);
		d_rses[f] = sinf(2*PI/period);
		d_dampes[f] = powf(1.0/PI, 1.0/period);
	}
}

extern "C"
void sp_init(int _bufFrameLength, int _freqCount, float minFreq, float maxFreq, float rate) {
	bufFrameLength = _bufFrameLength;
	freqCount = _freqCount;

	CUT_DEVICE_INIT();

	CUDA_SAFE_CALL( hipMalloc((void **)&d_data, bufFrameLength * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_freq, freqCount * bufFrameLength * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_ifreq, freqCount * bufFrameLength * sizeof(int)) );

	CUDA_SAFE_CALL( hipMalloc((void **)&d_xs, freqCount * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_ys, freqCount * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_as, freqCount * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_maxes, freqCount * sizeof(float)) );

	CUDA_SAFE_CALL( hipMalloc((void **)&d_periods, freqCount * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_rces, freqCount * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_rses, freqCount * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc((void **)&d_dampes, freqCount * sizeof(float)) );

	float *periods = (float*)malloc(sizeof(float) * freqCount);
	float max = rate/minFreq;
	float min = rate/maxFreq;
    min = logf(min);
    max = logf(max);

	for (int f=0; f<freqCount; f++){
      periods[f] = expf(min + f * (max - min) / (freqCount - 1));
	}

	CUDA_SAFE_CALL( hipMemcpy(d_periods, periods, freqCount * sizeof(float), hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	dev_init<<<BLOCK_N, THREAD_N>>>(freqCount, d_xs, d_ys, d_as, d_maxes, d_periods, d_rces, d_rses, d_dampes);

	CUT_CHECK_ERROR("dev_init() execution failed\n");
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

	free(periods);
}

__device__ void rotate(float &x, float &y, const float rc, const float rs, const float c) {
	const float xn = (x-c)*rc - y*rs;
	y = (x-c)*rs + y*rc;
	x = xn+c;
}

__global__ void dev_freq(int frames, float *data, float* freq, int freqCount,
						 float *d_xs,
						 float *d_ys,
						 float *d_as,
						 float *d_maxes,
						 
						 float *d_periods, 
						 float *d_rces,
						 float *d_rses,
						 float *d_dampes
) {
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;
	const int threadN = blockDim.x * gridDim.x;

	for (int f = tid; f<freqCount; f+=threadN){
		float x = d_xs[f];
		float y = d_ys[f];
		float a = d_as[f];
		float m = d_maxes[f];
		//float maxy = 0;
		//const float period = d_periods[f]/2;
		const float rc = d_rces[f];
		const float rs = d_rses[f];
		const float damp = d_dampes[f];

		float p = 0;
		for (int t=0; t<frames; t++, p++) {
		    __syncthreads();
			const float d = data[t];		
			//rotate(x, y, rc, rs, d);
			y += d;
			rotate(x, y, rc, rs, 0);
			y *= damp;
			//x *= damp;

			//maxy = fmaxf(maxy, fabsf(y));

			/*if (p>=period) {
				p = 0;
				a = maxy;
				maxy = 0;
			}*/

			//a = sqrtf((x-d)*(x-d) + y*y);
			a = sqrtf(x*x + y*y) / d_periods[f];

			freq[f + freqCount*t] = a;

			m = fmaxf(m, a);
		}

		d_xs[f] = x;
		d_ys[f] = y;
		d_as[f] = a;
		d_maxes[f] = m;
	}
}

__global__ void dev_ifreq(int lenght, float *freq, int* ifreq, float max){
	const int tid = blockDim.x * blockIdx.x + threadIdx.x;
	const int threadN = blockDim.x * gridDim.x;

	for (int f = tid; f<lenght; f+=threadN){
	    __syncthreads();
		int ifr = (int)(freq[f]/max*255.0); 
		ifreq[f] = ifr | (ifr << 8) | (ifr << 16) | 0xFF000000;
	}
}

extern "C"
void sp_freq(int frames, float *data, float* freq) {
	CUDA_SAFE_CALL( hipMemcpy(d_data, data, frames * sizeof(float), hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	dev_freq<<<BLOCK_N, THREAD_N>>>(frames, d_data, d_freq, freqCount, d_xs, d_ys, d_as, d_maxes, d_periods, d_rces, d_rses, d_dampes);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

//	printf("before memory copy\n");
	CUDA_SAFE_CALL( hipMemcpy(freq, d_freq, freqCount * frames * sizeof(float), hipMemcpyDeviceToHost) );	
}

extern "C"
void sp_ifreq(int lenght, float* freq, int* ifreq, float max) {
	CUDA_SAFE_CALL( hipMemcpy(d_freq, freq, lenght * sizeof(float), hipMemcpyHostToDevice) );

    CUDA_SAFE_CALL( hipDeviceSynchronize() );
	dev_ifreq<<<BLOCK_N, THREAD_N>>>(lenght, d_freq, d_ifreq, max);
    CUDA_SAFE_CALL( hipDeviceSynchronize() );

	CUDA_SAFE_CALL( hipMemcpy(ifreq, d_ifreq, lenght * sizeof(float), hipMemcpyDeviceToHost) );	
}

extern "C"
float sp_max() {
	float *maxes = (float*)malloc(sizeof(float) * freqCount);

	CUDA_SAFE_CALL( hipMemcpy(maxes, d_maxes, freqCount * sizeof(float), hipMemcpyDeviceToHost) );	

	float m = 0;
	for (int i=0; i<freqCount; i++) {
		m = fmaxf(m, maxes[i]);
	}
	
	free(maxes);

	return m;
}

extern "C"
void sp_close() {
	CUDA_SAFE_CALL( hipFree(d_data) );
	CUDA_SAFE_CALL( hipFree(d_freq) );
	CUDA_SAFE_CALL( hipFree(d_ifreq) );

	CUDA_SAFE_CALL( hipFree(d_xs) );
	CUDA_SAFE_CALL( hipFree(d_ys) );
	CUDA_SAFE_CALL( hipFree(d_as) );
	CUDA_SAFE_CALL( hipFree(d_maxes) );

	CUDA_SAFE_CALL( hipFree(d_periods) );
	CUDA_SAFE_CALL( hipFree(d_rces) );
	CUDA_SAFE_CALL( hipFree(d_rses) );
	CUDA_SAFE_CALL( hipFree(d_dampes) );
}
